#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<chrono>
#include <iostream>

using namespace std;

__global__ void sum(char *x) {
    int tid = threadIdx.x;
    x[tid] = (x[tid] + 1) % 256;
}


int main() {
    //int N = 32;
    //int nbytes = N * sizeof(int);
    char *dx = NULL, *hx = NULL;
    //// 申请显存
    //hipMalloc((void**)&dx, nbytes);

      int minbyte = 64;
      int maxbyte = 4 << 20;

      for (int i = minbyte; i<=maxbyte; i *= 2) {
        int nbytes = i;

        auto start = chrono::steady_clock::now();
        hipMalloc((void**)&dx, nbytes);
        hx = (char*)malloc(nbytes);

        for(int j=0;j<nbytes;j++) {
            hx[j] = j % 256;
        }


        hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

        // call GPU
        sum<<<1, nbytes>>>(dx);

        // let gpu finish
        hipDeviceSynchronize();


        hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);


        auto end = chrono::steady_clock::now();
        cout << "size(B): " << nbytes << ","
             << chrono::duration_cast<chrono::microseconds>(end - start).count()
             << ", us" << endl;

        // free(p);
      }


    //printf("\n");
    //hipFree(dx);
    //free(hx);
    return 0;
}

