#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define checkErrors(err) __cudaErrorCheck(err, __LINE__)
static inline void __cudaErrorCheck(hipError_t err, const int line) {
  char *str;
  if (err != hipSuccess) {
    hipDrvGetErrorName(err, (const char **)&str);
    printf("[CUDA error] %04d \"%s\" line %d\n", err, str, line);
  }
}

__global__ void sum(int *x, int *y, int *z) {
  int tid = threadIdx.x;
  x[tid] += 1;
}

hipDevice_t device;
size_t totalGlobalMem;
hipCtx_t _context;
int block_size = 32;
hipFunction_t function;
char module_file[] = "sum.ptx";
char kernel_name[] = "_Z3sumPiS_S_";

void cudaRegisterFatbin() {
  // cuda driver API初始化
  hipInit(0);
  hipCtxCreate(&_context, 0, device);
}

void loadKernelFunction() {
  hipModule_t module;
  hipError_t err;

  // hipModuleLoad直接加载ptx文件
  //  其他api还要hipModuleLoadData等
  checkErrors(hipModuleLoad(&module, module_file));

  checkErrors(hipModuleGetFunction(&function, module, kernel_name));
}

int main() {
  // cuda初始化
  cudaRegisterFatbin();
  loadKernelFunction();

  int N = 32;
  int nbytes = N * sizeof(int);
  // int i = 0;
  int *dx = NULL, *hx = NULL;
  // 申请显存
  // hipMalloc((void**)&dx, nbytes);
  checkErrors(hipMalloc((hipDeviceptr_t *)&dx, nbytes));

  // 申请成功
  if (dx == NULL) {
    printf("GPU alloc fail");
    return -1;
  }

  // 申请CPU内存
  hx = (int *)malloc(nbytes);
  if (hx == NULL) {
    printf("CPU alloc fail");
    return -1;
  }

  // init: hx: 0..31
  printf("hx original:\n");
  for (int i = 0; i < N; i++) {
    hx[i] = i;
    printf("%d ", hx[i]);
  }
  printf("\n");

  // copy to GPU
  // hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
  checkErrors(hipMemcpyHtoD((hipDeviceptr_t)dx, hx, nbytes));

  // call GPU
  // sum<<<1, N>>>(dx, dx ,dx);
  void **param = (void **)malloc(sizeof(void *) * 3);
  unsigned int sharedMemBytes = 0;
  hipStream_t hStream = 0;
  param[0] = &dx;
  param[1] = &dx;
  param[2] = &dx;
  checkErrors(hipModuleLaunchKernel(function, 1, 1, 1, 32, 1, 1, sharedMemBytes, hStream, param, NULL));

  // wait gpu to finish
  hipDeviceSynchronize();

  // copy data to host
  // hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);
  checkErrors(hipMemcpyDtoH(hx, (hipDeviceptr_t)dx, nbytes));

  printf("hx after:\n");
  for (int i = 0; i < N; i++) {
    printf("%d ", hx[i]);
  }

  printf("\n");
  hipFree(dx);
  free(hx);
  return 0;
}


