#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

typedef double FLOAT;
__global__ void sum(FLOAT *x, FLOAT *y, FLOAT *z) {
	int tid = threadIdx.x;
	x[tid] += 1;
}

int main() {
	int N = 32;
	int nbytes = N * sizeof(FLOAT);
	int i = 0;
	FLOAT *dx = NULL, *hx = NULL;
	// 申请显存
	hipMalloc((void**)&dx, nbytes);
	
	// 申请成功
	if (dx == NULL) {
		printf("GPU alloc fail");
		return -1;
	}

	// 申请CPU内存
	hx = (FLOAT*)malloc(nbytes);
	if (hx == NULL) {
		printf("CPU alloc fail");
		return -1;
	}

	// init: hx: 0..31
	printf("hx original:\n");
	for(int i=0;i<N;i++) {
		hx[i] = i;
		printf("%lf ", hx[i]);
	}
	printf("\n");

	// copy to GPU
	hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

	// call GPU
	sum<<<1, N>>>(dx, dx ,dx);

	// let gpu finish
	hipDeviceSynchronize();

	// copy data to CPU
	hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);

	printf("hx after:\n");
	for(int i=0;i<N;i++) {
		printf("%lf ", hx[i]);
	}
	printf("\n");
	hipFree(dx);
	free(hx);
	return 0;
}

