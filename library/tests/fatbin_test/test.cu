#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void sum(int *x, int *y, int *z) {
	int tid = threadIdx.x;
	x[tid] += 1;
}

int main() {
	int N = 32;
	int nbytes = N * sizeof(int);
	int *dx = NULL, *hx = NULL;
	// 申请显存
	hipMalloc((void**)&dx, nbytes);
	
	// 申请成功
	if (dx == NULL) {
		printf("GPU alloc fail");
		return -1;
	}

	// 申请CPU内存
	hx = (int*)malloc(nbytes);
	if (hx == NULL) {
		printf("CPU alloc fail");
		return -1;
	}

	// init: hx: 0..31
	printf("hx original:\n");
	for(int i=0;i<N;i++) {
		hx[i] = i;
		printf("%d ", hx[i]);
	}
	printf("\n");

	// copy to GPU
	hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

	// call GPU
	sum<<<1, N>>>(dx, dx ,dx);

	// let gpu finish
	hipDeviceSynchronize();

	// copy data to CPU
	hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);

	printf("hx after:\n");
	for(int i=0;i<N;i++) {
		printf("%d ", hx[i]);
	}
	printf("\n");
	hipFree(dx);
	free(hx);
	return 0;
}

