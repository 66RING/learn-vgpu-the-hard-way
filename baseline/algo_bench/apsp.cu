#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <stdlib.h>

#include <limits.h>
#include <unordered_map>
#include <unordered_set>
#include <chrono>
using namespace std;

constexpr int THREADS_PER_BLOCK = 1024;
constexpr int THREADS_PER_BLOCK_2D = 32;

typedef struct {
    int u;
    int v;
    int w;
} Edge;

int next_int (ifstream &ifs) {
    while (!isdigit(ifs.peek())) {
        ifs.get();
    }
    int output;
    ifs >> output;
    return output;
}

Edge* parse_file (const char *filename, int &nodes, int &edges) {
    ifstream ifs(filename);

    nodes = next_int(ifs);
    edges = next_int(ifs);

    Edge *out = (Edge*) malloc(edges*sizeof(Edge));
    for (int i = 0; i < edges; i++) {
        out[i].u = next_int(ifs);
        out[i].v = next_int(ifs);
        out[i].w = next_int(ifs);
    }
    return out;
}

__global__ void init_matrix (int *matrix, int num_nodes) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < num_nodes && col < num_nodes) {
        int index = row * num_nodes + col;
        matrix[index] = (row == col) ? 0 : INT_MAX;
    }
}

__global__ void fill_matrix (Edge *edge_list, int *matrix, int num_nodes, int num_edges) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_edges) {
        int midx = edge_list[index].u * num_nodes + edge_list[index].v;
        matrix[midx] = edge_list[index].w;
    }
}

__global__ void apsp (int *matrix, int num_nodes) {
    extern __shared__ int sdata[];
    int row = blockIdx.x;
    int col = blockIdx.y;
    if (row >= num_nodes || col >= num_nodes) {
        return;
    }

    int tid = threadIdx.x;
    int inter_node = blockIdx.z * blockDim.x + tid;
    int prev_cost = row * num_nodes + inter_node;
    int next_cost = inter_node * num_nodes + col;

    bool inf_cost = inter_node >= num_nodes ||
        matrix[prev_cost] == INT_MAX ||
        matrix[next_cost] == INT_MAX;
    sdata[tid] = inf_cost ? INT_MAX :
            matrix[prev_cost] + matrix[next_cost];
    __syncthreads();

    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] = min(sdata[tid], sdata[tid + stride]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicMin(&matrix[row * num_nodes + col], sdata[0]);
    }
}

int* list_to_matrix_cuda (Edge *h_edge_list, int num_nodes, int num_edges) {
    int in_bytes = num_edges*sizeof(Edge);
    int out_bytes = num_nodes*num_nodes*sizeof(int);
    Edge *d_edge_list;
    int *d_matrix;
    hipMalloc((void **) &d_edge_list, in_bytes);
    hipMalloc((void **) &d_matrix, out_bytes);
    hipMemcpy(d_edge_list, h_edge_list, in_bytes, hipMemcpyHostToDevice);
    

    dim3 dimBlock(THREADS_PER_BLOCK_2D, THREADS_PER_BLOCK_2D);
    int num_blocks_2d = (num_nodes + THREADS_PER_BLOCK_2D - 1) /
            (THREADS_PER_BLOCK_2D);
    dim3 dimGrid(num_blocks_2d, num_blocks_2d);
    init_matrix<<<dimGrid, dimBlock>>>(d_matrix, num_nodes);
    int num_blocks = (num_edges + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    fill_matrix<<<num_blocks, THREADS_PER_BLOCK>>>(d_edge_list, d_matrix, num_nodes, num_edges);

    hipFree(d_edge_list);
    return d_matrix;
}

int* apsp_cuda (Edge *h_edge_list, int num_nodes, int num_edges) {
    int *d_matrix = list_to_matrix_cuda(h_edge_list, num_nodes, num_edges);

    dim3 dimGrid(num_nodes, num_nodes,
            (num_nodes + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    for (int i = num_nodes; i > 0; i >>= 1) {
        apsp<<<dimGrid, THREADS_PER_BLOCK, THREADS_PER_BLOCK*sizeof(int)>>>(d_matrix, num_nodes);
    }

    int bytes = num_nodes*num_nodes*sizeof(int);
    int *h_matrix = (int*) malloc(bytes);
    hipMemcpy(h_matrix, d_matrix, bytes, hipMemcpyDeviceToHost);
    hipFree(d_matrix);
    return h_matrix;
}

int* list_to_matrix_serial (Edge *edge_list, int num_nodes, int num_edges) {
    int *matrix = (int*) malloc(num_nodes*num_nodes*sizeof(int));
    for (int row = 0; row < num_nodes; row++) {
        for (int col = 0; col < num_nodes; col++) {
            matrix[row * num_nodes + col] = (row == col) ? 0 : INT_MAX;
        }
    }

    for (int i = 0; i < num_edges; i++) {
        Edge e = edge_list[i];
        matrix[e.u * num_nodes + e.v] = e.w;
    }
    return matrix;
}

int* apsp_serial (Edge *edge_list, int num_nodes, int num_edges) {
    int *matrix = list_to_matrix_serial(edge_list, num_nodes, num_edges);

    for (int k = 0; k < num_nodes; k++) {
        for (int row = 0; row < num_nodes; row++) {
            for (int col = 0; col < num_nodes; col++) {
                int prev_path = row * num_nodes + k;
                int next_path = k * num_nodes + col;
                bool inf_dist = matrix[prev_path] == INT_MAX ||
                        matrix[next_path] == INT_MAX;
                int cost = inf_dist ? INT_MAX :
                    matrix[prev_path] + matrix[next_path];
                int index = row * num_nodes + col;
                matrix[index] = min(matrix[index], cost);
            }
        }
    }
    return matrix;
}

int main() {
    int num_nodes;
    int num_edges;
    cout << "Parsing Input File" << endl;
    Edge *edge_list = parse_file("graphs/dir_8.txt", num_nodes, num_edges);
    Edge *dummy = parse_file("graphs/dir_8.txt", num_nodes, num_edges);
    cout << "Parsed Input File" << endl << endl;
    
	{
		// Load kernel into GPU.
		apsp_cuda(dummy, num_nodes, num_edges);
		hipDeviceSynchronize();
	}
    cout << "Starting Parallel" << endl;
	auto start = chrono::steady_clock::now();
    int *matrix_p = apsp_cuda(edge_list, num_nodes, num_edges);
	hipDeviceSynchronize();
	auto end = chrono::steady_clock::now();
	cout << chrono::duration_cast<chrono::microseconds>(end - start).count() << ", us" << endl;
    cout << "Finished Parallel" << endl << endl;

    cout << "Starting Serial" << endl;
    int *matrix_s = apsp_serial(edge_list, num_nodes, num_edges);
    cout << "Finished Serial" << endl << endl;
    
    cout << "Checking Result" << endl;
    for (int i = 0; i < num_nodes*num_nodes; i++) {
        if (matrix_p[i] != matrix_s[i]) {
            cout << "FAILURE!!!" << endl;
            return 0;
        }
    }
    cout << "SUCCESS!!!" << endl;
    return 0;
}
